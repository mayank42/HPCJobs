#include "hip/hip_runtime.h"
#include "job2b.h"
int main(){
	int gridx,gridy;
	printf("Running 1D operation:\n");
	fflush(stdout);
	//Host arrays
	int *h_arr = (int*)malloc(oneDsize*sizeof(int));
	int h_oneDmask[5]={1,1,0,1,1};
	int *h_oneDres=(int*)malloc(oneDsize*sizeof(int));
	int a,b;
	for(a=0;a<oneDsize;a++)h_arr[a]=(int)(INPUT_MAX*(float)rand()/RAND_MAX);
	//Device arrays
	hipError_t err;
	int *d_arr;
	int *d_oneDmask;
	int *d_oneDres;
	err=hipMalloc((void**)&d_arr,oneDsize*sizeof(int));
	//debug
	//copy
	err=hipMemcpy(d_arr,h_arr,oneDsize*sizeof(int),hipMemcpyHostToDevice);
	//debug
	err=hipMalloc((void**)&d_oneDres,oneDsize*sizeof(int));
	//debug
	err=hipMalloc((void**)&d_oneDmask,5*sizeof(int));
	//debug
	//copy
	err=hipMemcpy(d_oneDmask,h_oneDmask,5*sizeof(int),hipMemcpyHostToDevice);
	
	dim3 block(1,64,1);
	gridy = (int)ceil(oneDsize/64.0);
	dim3 grid(1,gridy,1);
	printf("Convolving...");
	conv1D<<<grid,block>>>(d_arr,d_oneDmask,d_oneDres,oneDsize,5,2);
	printf("Done.\n");
	fflush(stdout);
	err=hipGetLastError();
	//debug
	//copy
	err=hipMemcpy(h_oneDres,d_oneDres,oneDsize*sizeof(int),hipMemcpyDeviceToHost);
	printf("Array:\n");
	for(a=0;a<oneDsize;a++)printf("%d ",h_arr[a]);
	printf("\n");
	printf("Convolution:\n");
	for(a=0;a<oneDsize;a++)printf("%d ",h_oneDres[a]);
	printf("\n");
	fflush(stdout);
	free(h_arr);
	free(h_oneDres);
	err=hipFree(d_arr);
	//debug
	hipFree(d_oneDres);
	//debug
	hipFree(d_oneDmask);
	//debug
	/***********************************************************************************
	*										   *
	*      TWO D CONVOLUTION							   *
	*										   *
	***********************************************************************************/
	int **h_A = (int**)malloc(twoDsize1*sizeof(int*));
	float **h_R = (float**)malloc(twoDsize1*sizeof(float*));
	float h_twoDmask[3][3] = {{0.125,0.125,0.125},{0.125,0.0,0.125},{0.125,0.125,0.125}};
	for(a=0;a<twoDsize1;a++)h_A[a]=(int*)malloc(twoDsize2*sizeof(int));
	for(a=0;a<twoDsize1;a++)h_R[a]=(float*)malloc(twoDsize2*sizeof(float));
	for(a=0;a<twoDsize1;a++){
		for(b=0;b<twoDsize2;b++){
			h_A[a][b]=(int)(INPUT_MAX*(float)rand()/RAND_MAX);
		}
	}
	int *d_A;
	float *d_R;
	float *d_twoDmask;
	err = hipMalloc((void**)&d_A,twoDsize1*twoDsize2*sizeof(int));
	//debug
	//copy
	for(a=0;a<twoDsize1;a++){
		err = hipMemcpy(d_A+a*twoDsize2,h_A[a],twoDsize2*sizeof(int),hipMemcpyHostToDevice);
		//debug
	}
	err = hipMalloc((void**)&d_R,twoDsize1*twoDsize2*sizeof(float));
	//debug
	//copy
	for(a=0;a<twoDsize1;a++){
		err = hipMemcpy(d_R+a*twoDsize2,h_R[a],twoDsize2*sizeof(float),hipMemcpyHostToDevice);
		//debug
	}
	err = hipMalloc((void**)&d_twoDmask,3*3*sizeof(float));
	//debug
	//copy
	for(a=0;a<3;a++){
		err = hipMemcpy(d_twoDmask+a*3,h_twoDmask[a],3*sizeof(float),hipMemcpyHostToDevice);
		//debug
	}
	printf("Orignal Matrix:\n");
	fflush(stdout);
	for(a=0;a<twoDsize1;a++){
		for(b=0;b<twoDsize2;b++){
			printf("%d ",h_A[a][b]);
		}
		printf("\n");
	}
	fflush(stdout);
	dim3 blockdim(8,8,1);
	gridx = (int)ceil(twoDsize1/8.0);
	gridy = (int)ceil(twoDsize2/8.0);
	dim3 griddim(gridx,gridy,1);
	printf("Convolving...");
	conv2D<<<griddim,blockdim>>>(d_A,d_twoDmask,d_R,twoDsize1,twoDsize2,3,3,1,1);
	printf("Done.\n");
	fflush(stdout);
	err = hipGetLastError();
	//debug
	for(a=0;a<twoDsize1;a++){
		err = hipMemcpy(h_R[a],d_R+a*twoDsize2,twoDsize2*sizeof(float),hipMemcpyDeviceToHost);
		//debug
	}
	printf("Convolution:\n");
	fflush(stdout);
	for(a=0;a<twoDsize1;a++){
		for(b=0;b<twoDsize2;b++){
			printf("%f ",h_R[a][b]);
		}
		printf("\n");
	}
	fflush(stdout);
	for(a=0;a<twoDsize1;a++){
		free(h_A[a]);
		free(h_R[a]);
	}
	free(h_A);
	free(h_R);
	err = hipFree(d_A);
	//debug
	err = hipFree(d_R);
	//debug
	err = hipFree(d_twoDmask);
	//debug
	err = hipDeviceReset();
	//debug
	printf("Last err: %d\n",err);
	return 0;
}	
	
	
