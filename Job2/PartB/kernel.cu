
#include <hip/hip_runtime.h>
__global__ void conv1D(int *arr,int *mask,int *res,int n,int m,int c){
	int idi = blockIdx.y*blockDim.y+threadIdx.y;
	if(idi<n){
		res[idi]=0;
		int a,b;
		b=idi-c;
		for(a=0;a<m;a++,b++){
			if(b>=0 && b<n){
				res[idi]+=mask[a]*arr[b];
			}
		}
	}
}
__global__ void conv2D(int *arr,float *mask,float *res,int n1,int n2, int m1,int m2,int c1,int c2){
	int idi = blockIdx.y*blockDim.y+threadIdx.y;
	int idj = blockIdx.x*blockDim.x+threadIdx.x;
	if(idi<n1 && idj<n2){
		int a,b,c,d;
		c=idi-c1;
		res[idi*n1+idj]=0.0;
		for(a=0;a<m1;a++,c++){
			d=idj-c2;
			for(b=0;b<m2;b++,d++){
				if(c>=0 && c<n1 && d>=0 && d<n2){
					res[idi*n1+idj]+=mask[a*m1+b]*arr[c*n1+d];
				}
			}
		}
	}
}
