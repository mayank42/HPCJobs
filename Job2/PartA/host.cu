#include "hip/hip_runtime.h"
#include "job2a.h"
int main(){
	int **h_A = (int**)malloc(size*sizeof(int*));
	int **h_R = (int**)malloc(size*sizeof(int*));
	int a,b;
	hipError_t err;
	for(a=0;a<size;a++)h_A[a]=(int*)malloc(size*sizeof(int));
	for(a=0;a<size;a++)h_R[a]=(int*)malloc(size*sizeof(int));
	for(a=0;a<size;a++){
		for(b=0;b<size;b++){
			h_A[a][b]=rand();
		}
	}
	int * d_A;
	err = hipMalloc((void**)&d_A,size*size*sizeof(int));
	debug(err,"Unable to allocate matrix on device")
	//copy
	for(a=0;a<size;a++){
		err = hipMemcpy(d_A+a*size,h_A[a],size*sizeof(int),hipMemcpyHostToDevice);
		debug(err,"Failed to copy to device")
	}
	printf("Orignal Matrix:\n");
	fflush(stdout);
	for(a=0;a<size;a++){
		for(b=0;b<size;b++){
			printf("%d ",h_A[a][b]);
		}
		printf("\n");
	}
	fflush(stdout);
	dim3 blockdim(8,8,1);
	int gridx = (int)ceil(8.0/size);
	dim3 griddim(gridx,gridx,1);
	printf("Swapping...");
	swap<<<griddim,blockdim>>>(d_A,size);
	printf("Done.\n");
	fflush(stdout);
	err = hipGetLastError();
	debug(err,"Last error in execution")
	for(a=0;a<size;a++){
		err = hipMemcpy(h_R[a],d_A+a*size,size*sizeof(int),hipMemcpyDeviceToHost);
		debug(err,"Failed to copy back to host")
	}
	printf("Swapped array:\n");
	for(a=0;a<size;a++){
		for(b=0;b<size;b++){
			printf("%d ",h_R[a][b]);
		}
		printf("\n");
	}
	fflush(stdout);
	for(a=0;a<size;a++){
		free(h_A[a]);
		free(h_R[a]);
	}
	free(h_A);
	free(h_R);
	err = hipFree(d_A);
	debug(err,"Failed to free memory on device")
	err = hipDeviceReset();
	debug(err,"Unable to reset device")
	printf("Last err: %d\n",err);
	return 0;
}

		
	
	
