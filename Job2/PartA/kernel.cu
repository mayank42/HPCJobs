
#include <hip/hip_runtime.h>
__global__ void swap(int *A,int n){
	int idi = blockIdx.y*blockDim.y+threadIdx.y;
	int idj = blockIdx.x*blockDim.x+threadIdx.x;
	if(idi<n && idj<=idi){
		if(idj%2==0 && idj<n-1){
			int temp=A[idi*n+idj];
			A[idi*n+idj]=A[idi*n+idj+1];
			A[idi*n+idj+1]=temp;
		}
		int temp=idi;
		idi=idj;
		idj=temp;
		if(idi!=idj && idj%2==0 && idj<n-1){
			int temp=A[idi*n+idj];
			A[idi*n+idj]=A[idi*n+idj+1];
			A[idi*n+idj+1]=temp;
		}
		__syncthreads();
		temp=A[idi*n+idj];
		A[idi*n+idj]=A[idj*n+idi];
		A[idj*n+idi]=temp;
	}
}
