#include<iostream>
#include<string>
#include<hip/hip_runtime.h>
using namespace std;
int main(){
	struct hipDeviceProp_t prop;
	hipError_t err;
	err = hipGetDeviceProperties(&prop,0);
	if(err!=hipSuccess){
		cout<<"Get failed. Exiting."<<endl;
	}
	else{
		cout<<"Name   :   "<<string(prop.name)<<endl;
		cout<<"Total global memory   :   "<<prop.totalGlobalMem/(1024*1024*1024.0)<<" GB"<<endl;
        	cout<<"Shared memmory per block   :   "<<prop.sharedMemPerBlock/(1024.0)<<" KB"<<endl;
	        cout<<"32 bit registers per block   :   "<<prop.regsPerBlock<<endl;
	        cout<<"Warp size (in threads)   :   "<<prop.warpSize<<endl;
	        cout<<"Max pitch allowed by mem copy   :   "<<prop.memPitch/(1024*1024*1024.0)<<" GB"<<endl;
	        cout<<"Max threads per block   :   "<<prop.maxThreadsPerBlock<<endl;
	        cout<<"Max thread dimensions   :   "<<"("<<prop.maxThreadsDim[0]<<","<<prop.maxThreadsDim[1]<<","<<prop.maxThreadsDim[2]<<")"<<endl;
		cout<<"Max grid dimensions   :   "<<"("<<prop.maxGridSize[0]<<","<<prop.maxGridSize[1]<<","<<prop.maxGridSize[2]<<")"<<endl;
	        cout<<"Max const memory   :   "<<prop.totalConstMem/1024.0<<" KB"<<endl;
	        cout<<"Major compute capability   :   "<<prop.major<<endl;
	        cout<<"Minor compute capability   :   "<<prop.minor<<endl;
	        cout<<"Clock frequency   :   "<<prop.clockRate/1000.0<<" MHz"<<endl;
	        cout<<"Alignment requirement for textures   :   "<<prop.textureAlignment<<endl;
	        cout<<"Device can concurrently copy memory and execute a kernel   :   "<<(bool)prop.deviceOverlap<<endl;
	        cout<<"Number of multiprocessors on device   :   "<<prop.multiProcessorCount<<endl;
	        cout<<"Specified whether there is a run time limit on kernels   :   "<<(bool)prop.kernelExecTimeoutEnabled<<endl;
	        cout<<"Integrated   :   "<<(bool)prop.integrated<<endl;
	        cout<<"Can map host memory   :   "<<(bool)prop.canMapHostMemory<<endl;
	        cout<<"Compute Mode   :   "<<prop.computeMode<<endl;
	        cout<<"Concurrent kernels   :   "<<(bool)prop.concurrentKernels<<endl;
	        cout<<"ECC support   :   "<<(bool)prop.ECCEnabled<<endl;
	        cout<<"PCI bus id   :   "<<prop.pciBusID<<endl;
	        cout<<"PCI device id   :   "<<prop.pciDeviceID<<endl;
	        cout<<"TCC Driver   :   "<<(bool)prop.tccDriver<<endl;
	}
	return 0;
}
