#include "hip/hip_runtime.h"
#include "MatRedux.h"
int main(int argc,char *argv[]){
	tab=TAB0;
	MSG("Checking data file");
	ifstream data(FILE_PATH,ios::binary);
	if(data.is_open()){
		ACK(OK);
	}
	else{
		ACK(FAIL);
		DEBUG("Unable to open file.");
	}
	
	size_t length;
	data.read(reinterpret_cast<char*>(&length),sizeof(length));
	POST("Number of 2x2 matrices",length);
	int flag;
	data.read(reinterpret_cast<char*>(&flag),sizeof(flag));
	if(flag){
		POST("Alignment","Column Major");
	}
	else{
		 POST("Alignment","Row Major");
	}

	POST("Batch volume percentage",VOLUME);
	size_t bsize = (size_t)(length*VOLUME);
	if(bsize%1024!=0){
		POST("Batch size unaligned. Rounding to nearest multiple of 1024","");
		bsize+=1024-bsize%1024;
	}
	POST("Batch size",bsize);
	POST("Block dimension",BLOCKX);
	MSG("Calibrating grid dimension");
	unsigned long long gridx = (unsigned long long)(bsize/1024);
	if(gridx>2147483647){
		ACK(FAIL);
		DEBUG("Batch volume too large. Reduce it");
	}
	ACK(OK);
	POST("Grid dimension",gridx);
	dim3 grid(gridx,1,1);
	dim3 block(BLOCKX,1,1);

	
	size_t bstart = 0;
	int count = 1;
	vector<float> bans;
	hipError_t err;
	clock_t begin,end;
	double cudaTotal,serialTotal;
	cudaTotal=serialTotal=0.0;
	while(bstart<length){
		POST("Processing batch count",count);		
		tab=TAB1;

		/*******************HOST MEMORY ALLOC*******************/
		MSG("Allocating memory on host");
		float *h_mat;
		try{
			h_mat = new float[MAT_SIZE*bsize];
		}catch(exception &e){
			ACK(FAIL);
			DEBUG(string("Allocation error. ")+e.what());
		};
		ACK(OK);
		/*******************HOST MEMORY ALLOC*******************/


		/*******************DATA FILE READ*******************/
		MSG("Reading data from file");
		try{
			if(bstart+bsize<length)
				data.read(reinterpret_cast<char*>(h_mat),MAT_SIZE*bsize*sizeof(float));
			else{
				data.read(reinterpret_cast<char*>(h_mat),MAT_SIZE*(length-bstart)*sizeof(float));
				for(size_t loopvar = (length-bstart)*MAT_SIZE;loopvar<MAT_SIZE*bsize;++loopvar)h_mat[loopvar]=0.0;
			}
		}catch(exception &e){
			ACK(FAIL);
			DEBUG(string("Data read failed. ")+e.what());
		}
		ACK(OK);
		/*******************DATA FILE READ*******************/

		
		/*******************DEVICE MEMORY ALLOC*******************/		
		float *d_imat;
		float *d_omat;
		MSG("Allocating memory on device");
		err = hipMalloc(&d_imat,MAT_SIZE*bsize*sizeof(float));
		CDEBUG("Device memory allocation",err);
		MSG("Allocating memory on device");
		err = hipMalloc(&d_omat,MAT_SIZE*bsize/1024*sizeof(float));
		CDEBUG("Device memory allocation",err);
		/*******************DEVICE MEMORY ALLOC*******************/

		
		/*******************DEVICE MEM COPY*******************/
		MSG("Copying from host to device");
		err = hipMemcpy(d_imat,h_mat,MAT_SIZE*bsize*sizeof(float),hipMemcpyHostToDevice);
		CDEBUG("Host to device memcpy",err);
		/*******************DEVICE MEM COPY*******************/		

		
		/*******************REDUCTION*******************/
		POST("Starting reduction","");
		tab=TAB2;
		if(flag){
				//colRedux(grid,block,d_imat,d_omat,h_mat,bsize,bans);
			err = hipGetLastError();
			CDEBUG("Column kernel reduction",err);
		}
		else{
			err = rowRedux(grid,block,d_imat,d_omat,bsize,bans,&cudaTotal);
			MSG("Reduction status");
			CDEBUG("Row kernel reduction",err);
		}
		tab=TAB1;
		/*******************REDUCTION*******************/
		

		/*******************MEMORY FREE*******************/
		MSG("Freeing host memory");
		delete[] h_mat;
		ACK(OK);
		MSG("Freeing device input memory");
		err = hipFree(d_imat);
		CDEBUG("Free device memory",err);
		MSG("Freeing device output memory");
		err = hipFree(d_omat);
		CDEBUG("Free device memory",err);
		/*******************MEMORY FREE*******************/

		
		bstart+=bsize;
		count++;
	}
	tab=TAB0;	

	MSG("Adding results on host");
	vector<float> ans(4,0.0);
	begin = clock();
	addMat(bans,ans,flag);
	end = clock();
	cout<<begin<<" "<<end<<endl;
	cudaTotal+=(double)(end-begin);
	ACK("[  Done  ]");
	
	
	data.close();	
	/*******************TESTING*******************/
	MSG("Testing");
	string comment;
	if(sumTest(ans,comment,&serialTotal)){
		ACK(OK);
		cout<<setprecision(8);
		POST("Cuda time",cudaTotal);
		POST("Serial time",serialTotal*1000/CLOCKS_PER_SEC);
	}
	else{
		ACK(FAIL);
		POST("Fail message",comment);
	}
	/*******************TESTING*******************/
	
	MSG("Resetting device");
	err = hipDeviceReset();
	CDEBUG("Device reset",err);
	return 0;
}

bool sumTest(vector<float> &ans,string &comm,double *tot){
	ifstream data(FILE_PATH,ios::binary);
	if(!data.is_open()){
		comm = "Unable to open data file.";
		return false;
	}
	size_t length;
	int flag;
	data.read(reinterpret_cast<char*>(&length),sizeof(length));
	data.read(reinterpret_cast<char*>(&flag),sizeof(flag));
	vector<float> res(4,0.0);
	if(!flag){
		float *mat = new float[1024*MAT_SIZE];
		int reads = 1024;
		clock_t end,begin;
		begin = clock();
		for(size_t start=0;start<length;start+=1024){
			if(start+1024>length)reads = length-start;
			data.read(reinterpret_cast<char*>(mat),reads*MAT_SIZE*sizeof(float));
			begin=clock();
			for(int a=0;a<reads;a++){
				res[0]+=mat[a*4];
				res[1]+=mat[a*4+1];
				res[2]+=mat[a*4+2];
				res[3]+=mat[a*4+3];
			}
			end=clock();
			*tot = *tot + (double)(end-begin);
		}
	}
	if(ans[0]==res[0] && ans[1]==res[1] && ans[2]==res[2] && ans[3]==res[3]){
		comm="Test successful.";
		return true;
	}
	else{
		stringstream com;
		com<<"Test unsuccessful.\nSerial result:\n";
		com<<setprecision(8);
		com<<res[0]<<" "<<res[1]<<"\n"<<res[2]<<" "<<res[3]<<"\nCuda result:\n";
		com<<ans[0]<<" "<<ans[1]<<"\n"<<ans[2]<<" "<<ans[3];
		comm=com.str();

		return false;
	}
}

hipError_t rowRedux(dim3 &grid,dim3 &block,float *d_imat,float *d_omat,size_t length,vector<float> &bans,double *tot){
	float *arr[]={d_imat,d_omat};
	int pos=0;
	int sync_len=0;
	hipError_t err;
	POST("Reducing on length",length);
	while(length>SYNC_LIM){
		length/=1024;
		sync_len++;
	}
	POST("Sync length",sync_len);
	float ctime;
	for(int a=0;a<sync_len;a++){
		POST("Iteration number",a+1);
		hipEvent_t start,stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start,0);
		row_kernel<<<grid,block>>>(arr[pos%2],arr[(pos+1)%2]);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&ctime, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);
		cout<<"Here"<<ctime<<endl;
		*tot=*tot+(double)ctime;
		err = hipGetLastError();
		if(err!=hipSuccess)return err;
		pos++;
	}
	return getRowResult(arr[pos%2],length,bans);
}
hipError_t getRowResult(float *d_omat,size_t length,vector<float> &bans){
	float *h_mat = new float[MAT_SIZE*length];
	hipError_t err;
	err = hipMemcpy(h_mat,d_omat,MAT_SIZE*length*sizeof(float),hipMemcpyDeviceToHost);
	if(err!=hipSuccess)return err;
	float a,b,c,d;
	a=b=c=d=0.0;
	for(int i=0;i<length;++i){
		a+=h_mat[MAT_SIZE*i];
		b+=h_mat[MAT_SIZE*i+1];
		c+=h_mat[MAT_SIZE*i+2];
		d+=h_mat[MAT_SIZE*i+3];
	}
	bans.push_back(a);
	bans.push_back(b);
	bans.push_back(c);
	bans.push_back(d);
	return hipSuccess;
}
void addMat(vector<float> &bans,vector<float> &ans,int flag){
	int l = bans.size()/4;
	ans[0]=ans[1]=ans[2]=ans[3]=0.0;
	for(int a=0;a<l;a++){
		ans[0]+=bans[a*4];
		ans[1]+=bans[a*4+1];
		ans[2]+=bans[a*4+2];
		ans[3]+=bans[a*4+3];
	}
}
