#include "MatRedux.h"
int main(int argc,char *argv[]){
	MSG("Checking data file");
	ifstream data(FILE_PATH,ios::binary);
	if(!data.is_open())ACK(OK);
	else{
		ACK(FAIL);
		DEBUG("Unable to open file.");
	}
	
	size_t length;
	data.read(reinterpret_cast<char*>(&length),sizeof(length));
	POST("Number of 2x2 matrices",length);
	int flag;
	data.read(reinterpret_cast<char*>(&flag),sizeof(flag));
	if(flag)POST("Alignment","Column Major");
	else POST("Alignment","Row Major");

	POST("Batch volume percentage",VOLUME);
	size_t bsize = (size_t)(length*VOLUME);
	if(bsize%1024!=0){
		POST("Batch size unaligned. Rounding to nearest multiple of 1024","");
		bsize+=1024-bsize%1024;
	}
	POST("Batch size",bsize);
	POST("Block dimension",BLOCKX);
	MSG("Calibrating grid dimension");
	unsigned long long gridx = (unsigned long long)(bsize/1024);
	if(gridx>2147483647){
		ACK(FAIL);
		DEBUG("Batch volume too large. Reduce it");
	}
	ACK(OK);
	POST("Grid dimension",gridx);
	dim3 grid(gridx,1,1);
	dim3 block(BLOCKX,1,1);

	
	size_t bstart = 0;
	int count = 1;
	vector<float> bans;
	hipError_t err;
	while(bstart<length){
		POST("Processing batch count",count);		

		/*******************HOST MEMORY ALLOC*******************/
		MSG("Allocating memory on host");
		try{
			float *h_mat = new float[MAT_SIZE*bsize];
		}catch(exception &e){
			ACK(FAIL);
			DEBUG(string("Allocation error. ")+e.what());
		};
		ACK(OK);
		/*******************HOST MEMORY ALLOC*******************/


		/*******************DATA FILE READ*******************/
		MSG("Reading data from file");
		try{
			if(bstart+bsize<length)
				data.read(reinterpret_cast<char*>(h_mat),MAT_SIZE*bsize*sizeof(float));
			else{
				data.read(reinterpret_cast<char*>(h_mat),MAT_SIZE*(length-bstart)*sizeof(float));
				for(size_t loopvar = (length-bstart)*MAT_SIZE;loopvar<MAT_SIZE*bsize;++loopvar)h_mat[loopvar]=0.0;
			}
		}catch(exception &e){
			ACK(FAIL);
			DEGBUG(string("Data read failed. ")+e.what());
		}
		ACK(OK);
		/*******************DATA FILE READ*******************/

		
		/*******************DEVICE MEMORY ALLOC*******************/		
		float *d_imat
		MSG("Allocating memory on device");
		err = hipMalloc(&d_imat,MAT_SIZE*bsize*sizeof(float));
		CDEBUG("Device memory allocation",err);
		err = hipMalloc(&d_omat,MAT_SIZE*bsize/1024*sizeof(float));
		CDEBUG("Device memory allocation",err);
		/*******************DEVICE MEMORY ALLOC*******************/

		
		/*******************DEVICE MEM COPY*******************/
		MSG("Copying from host to device");
		err = hipMemcpy(d_imat,h_mat,MAT_SIZE*bsize*sizeof(float),hipMemcpyHostToDevice);
		CDEBUG("Host to device memcpy",err);
		/*******************DEVICE MEM COPY*******************/		

		
		/*******************REDUCTION*******************/
		MSG("Reducing");
		if(flag){
			colRedux(grid,block,d_imat,d_omat,h_mat,bsize,bans);
			err = hipGetLastError();
			CDEBUG("Column kernel reduction",err);
		}
		else{
			rowRedux(grid,block,d_imat,d_omat,h_mat,bsize,bans);
			err = hipGetLastError();
			CDEBUG("Row kernel reduction",err);
		}
		/*******************REDUCTION*******************/
		

		/*******************MEMORY FREE*******************/
		MSG("Freeing host memory");
		delete[] h_mat;
		ACK(OK);
		MSG("Freeing device memory");
		err = hipFree(d_mat);
		CDEBUG("Free device memory",err);
		/*******************MEMORY FREE*******************/

		
		bstart+=bsize;
		count++;
	}
	
	MSG("Adding results on host");
	vector<float> ans;
	addMat(bans,ans,flag);
	ACK("[  Done  ]");
	
	
	data.close();	
	/*******************TESTING*******************/
	MSG("Testing");
	string comment;
	if(sumTest(ans,comment))
		ACK(OK);
	else{
		ACK(FAIL);
		POST("Fail message",comment);
	}
	/*******************TESTING*******************/
	
	MSG("Resetting device");
	err = hipDeviceReset();
	CDEBUG("Device reset",err);
	return 0;
}

bool sumTest(vector<float> &ans,string &comm){
	ifstream data(FILE_PATH,ios::binary);
	if(!data.is_open()){
		comm = "Unable to open data file.";
		return false;
	}
	size_t length;
	int flag;
	data.read(reinterpret_cast<char*>(&length),sizeof(length));
	data.read(reinterpret_cast<char*>(&flag),sizeof(flag));
	if(!flag){
		float *mat = new float[1024*MAT_SIZE];
		int reads = 1024;
		vector<float> res(4,0.0);
		for(size_t start=0;start<length;start+=1024){
			if(start+1024>length)reads = length-start;
			data.read(reinterpret_cast<char*>(mat),reads*MAT_SIZE*sizeof(float));
			for(int a=0;a<reads;a++){
				res[0]+=mat[a*4];
				res[1]+=mat[a*4+1];
				res[2]+=mat[a*4+2];
				res[3]+=mat[a*4+3];
			}
		}
	}
	if(ans[0]==res[0] && ans[1]==res[1] && ans[2]==res[2] && ans[3]==res[3]){
		comment="Test successful.";
		return true;
	}
	else{
		comment="Test unsuccessful.";
		return false;
	}
}

hipError_t rowRedux(dim3 &grid,dim3 &block,float *d_imat,float *d_omat,size_t length,vector<float> &bans){
	float *arr[]={d_imat,d_omat};
	int pos=0;
	int sync_len=0;
	while(length>SYNC_LIM){
		length/=1024;
		sync_len++;
	}
	for(int a=0;a<sync_len;a++){
		row_kernel<<<grid,block>>>(arr[pos%2],arr[(pos+1)%2]);
		err = hipGetLastError();
		if(err!=hipSuccess)return err;
		pos++;
	}
	return getRowResult(arr[pos%2],length,bans);
}
hipError_t getRowResult(float *d_omat,size_t length,vector<float> &bans){
	float *h_mat = new float[MAT_SIZE*length];
	hipError_t err;
	err = hipMemcpy(h_mat,d_omat,MAT_SIZE*length*sizeof(float),hipMemcpyDeviceToHost);
	if(err!=hipSuccess)return err;
	float a,b,c,d;
	a=b=c=d=0.0;
	for(int i=0;i<length;++i){
		a+=h_mat[MAT_SIZE*i];
		b+=h_mat[MAT_SIZE*i+1];
		c+=h_mat[MAT_SIZE*i+2];
		d+=h_mat[MAT_SIZE*i+3];
	}
	bans.push_back(a);
	bans.push_back(b);
	bans.push_back(c);
	bans.push_back(d);
	return hipSuccess;
}
