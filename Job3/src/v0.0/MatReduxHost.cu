#include "matRedux.h"
int main(int argc,char *argv[]){
	tab=TAB0;
	MSG("Checking data file");
	ifstream data(FILE_PATH,ios::binary);
	if(data.is_open()){
		ACK(OK);
	}
	else{
		ACK(FAIL);
		DEBUG("Unable to open file.");
	}
	
	size_t length;
	data.read(reinterpret_cast<char*>(&length),sizeof(length));
	POST("Number of 2x2 matrices",length);
	int flag;
	data.read(reinterpret_cast<char*>(&flag),sizeof(flag));
	if(flag){
		POST("Alignment","Column Major");
	}
	else{
		 POST("Alignment","Row Major");
	}
	POST("Batch volume percentage",VOLUME);
	size_t bsize = (size_t)(length*VOLUME);
	if(bsize%1024!=0){
		POST("Batch size unaligned. Rounding to nearest multiple of 1024","");
		bsize+=1024-bsize%1024;
	}
	POST("Batch size",bsize);
	POST("Block dimension",BLOCKX);
	MSG("Calibrating grid dimension");
	unsigned long long gridx = (unsigned long long)(bsize/1024);
	if(gridx>2147483647){
		ACK(FAIL);
		DEBUG("Batch volume too large. Reduce it");
	}
	ACK(OK);
	POST("Grid dimension",gridx);
	dim3 grid(gridx,1,1);
	dim3 block(BLOCKX,1,1);

	
	size_t bstart = 0;
	int count = 1;
	vector<double> bans;
	hipError_t err;
	float ctime;
	double diff;
	hipEvent_t start,stop;
	double cudaRedux,cudaMem,serialTotal;
	cudaRedux=cudaMem=serialTotal=0.0;
	while(bstart<length){
		POST("Processing batch count",count);		
		tab=TAB1;

		/*******************HOST MEMORY ALLOC*******************/
		MSG("Allocating memory on host");
		float *h_mat_f;
		double *h_mat;
		try{
			h_mat_f = new float[MAT_SIZE*bsize];
			h_mat = new double[MAT_SIZE*bsize];
			
		}catch(exception &e){
			ACK(FAIL);
			DEBUG(string("Allocation error. ")+e.what());
		};
		ACK(OK);
		/*******************HOST MEMORY ALLOC*******************/


		/*******************DATA FILE READ*******************/
		MSG("Reading data from file");
		try{
			if(bstart+bsize<=length)
				data.read(reinterpret_cast<char*>(h_mat_f),MAT_SIZE*bsize*sizeof(float));
			else{
				data.read(reinterpret_cast<char*>(h_mat_f),MAT_SIZE*(length-bstart)*sizeof(float));
				for(size_t loopvar = (length-bstart)*MAT_SIZE;loopvar<MAT_SIZE*bsize;++loopvar)h_mat_f[loopvar]=0.0f;
			}
		}catch(exception &e){
			ACK(FAIL);
			DEBUG(string("Data read failed. ")+e.what());
		}
		ACK(OK);
		if(!flag)
			for(size_t a=0;a<MAT_SIZE*bsize;a++)h_mat[a]=h_mat_f[a];
		else
			for(size_t a=0;a<bsize;++a){
				for(int b=0;b<MAT_SIZE;++b){
					h_mat[b*MAT_SIE+a] = h_mat_f[a*4+b];
				}
			}
		delete[] h_mat_f;
		/*******************DATA FILE READ*******************/

		
		/*******************DEVICE MEMORY ALLOC*******************/		
		double *d_imat;
		double *d_omat;
		MSG("Allocating memory on device");
		err = hipMalloc(&d_imat,MAT_SIZE*bsize*sizeof(double));
		CDEBUG("Device memory allocation",err);
		MSG("Allocating memory on device");
		err = hipMalloc(&d_omat,MAT_SIZE*(bsize/1024+1024)*sizeof(double));
		CDEBUG("Device memory allocation",err);
		/*******************DEVICE MEMORY ALLOC*******************/

		
		/*******************DEVICE MEM COPY*******************/
		MSG("Copying from host to device");
		CEVENTSET(start,stop,ctime);
		err = hipMemcpy(d_imat,h_mat,MAT_SIZE*bsize*sizeof(double),hipMemcpyHostToDevice);
		CEVENTGET(start,stop,ctime);
		cudaMem+=ctime;
		CDEBUG("Host to device memcpy",err);
		/*******************DEVICE MEM COPY*******************/		

		
		/*******************REDUCTION*******************/
		POST("Starting reduction","");
		tab=TAB2;
		if(flag){
			err = colRedux(grid,block,d_imat,d_omat,bsize,bans,flag,&cudaRedux,&cudaMem);
			err = hipGetLastError();
			CDEBUG("Column kernel reduction",err);
		}
		else{
			err = rowRedux(grid,block,d_imat,d_omat,bsize,bans,flag,&cudaRedux,&cudaMem);
			MSG("Reduction status");
			CDEBUG("Row kernel reduction",err);
		}
		tab=TAB1;
		/*******************REDUCTION*******************/
		

		/*******************MEMORY FREE*******************/
		MSG("Freeing host memory");
		delete[] h_mat;
		ACK(OK);
		MSG("Freeing device input memory");
		err = hipFree(d_imat);
		CDEBUG("Free device memory",err);
		MSG("Freeing device output memory");
		err = hipFree(d_omat);
		CDEBUG("Free device memory",err);
		/*******************MEMORY FREE*******************/

		
		bstart+=bsize;
		count++;
		tab = TAB0;
	}
	tab=TAB0;	

	MSG("Adding results on host");
	vector<double> ans(4,0.0);
	push_clock();
	addMat(bans,ans,flag);
	diff = push_clock();
	cudaRedux+=diff;
	ACK("[  Done  ]");
	clear_clock();
	
	
	data.close();	
	/*******************TESTING*******************/
	MSG("Testing");
	string comment;
	if(sumTest(ans,comment,&serialTotal)){
		ACK(OK);
		cout<<setprecision(8);
		stringstream cudaDisp;
		cudaDisp<<setprecision(8)<<cudaRedux<<" ( + "<<cudaMem<<" memcpy )";
		POST("Cuda time (ms)",cudaDisp.str());
		POST("Serial time (ms)",serialTotal*1000/CLOCKS_PER_SEC);
	}
	else{
		ACK(FAIL);
		POST("Fail message",comment);
	}
	/*******************TESTING*******************/
	
	MSG("Resetting device");
	err = hipDeviceReset();
	CDEBUG("Device reset",err);
	return 0;
}

bool sumTest(vector<double> &ans,string &comm,double *tot){
	ifstream data(FILE_PATH,ios::binary);
	if(!data.is_open()){
		comm = "Unable to open data file.";
		return false;
	}
	size_t length;
	int flag;
	double ctime;
	data.read(reinterpret_cast<char*>(&length),sizeof(length));
	data.read(reinterpret_cast<char*>(&flag),sizeof(flag));
	vector<double> res(4,0.0);
	double *acc = new double[4];
	size_t readBuf = VOLUME*length;
	size_t readCount = 0;
	size_t reads;
	if(!flag){
		float *mat_f = new float[MAT_SIZE*readBuf];
		double *mat = new double[MAT_SIZE*readBuf];
		while(readCount<length){
			if(readCount+readBuf<=length)
				reads = readBuf;
			else
				reads = length-readCount;
			data.read(reinterpret_cast<char*>(mat_f),reads*MAT_SIZE*sizeof(float));
			for(size_t a=0;a<MAT_SIZE*reads;++a)mat[a] = mat_f[a];
			push_clock();	
			lowRedux(mat,reads,acc,flag);		
			res[0]+=acc[0];
			res[1]+=acc[1];
			res[2]+=acc[2];
			res[3]+=acc[3];
			ctime = push_clock();
			*tot = *tot + ctime;
			readCount+=reads;
		}
		delete[] mat_f;
		delete[] mat;
	}
	clear_clock();
	data.close();
	delete[] acc;
	if(ans[0]==res[0] && ans[1]==res[1] && ans[2]==res[2] && ans[3]==res[3]){
		comm="Test successful.";
		return true;
	}
	else{
		stringstream com;
		com<<"Test unsuccessful.\nSerial result:\n";
		com<<setprecision(8);
		com<<res[0]<<" "<<res[1]<<"\n"<<res[2]<<" "<<res[3]<<"\nCuda result:\n";
		com<<ans[0]<<" "<<ans[1]<<"\n"<<ans[2]<<" "<<ans[3];
		comm=com.str();

		return false;
	}
}

hipError_t rowRedux(dim3 grid,dim3 block,double *d_imat,double *d_omat,size_t length,vector<double> &bans,int flag,double *rtot,double *mtot){
	double *arr[]={d_imat,d_omat};
	int pos=0;
	hipError_t err;
	POST("Reducing on length",length);
	POST("Sync length",SYNC_LEN);
	float ctime;
	double diff;
	hipEvent_t start,stop;
	unsigned int lag;
	int a=0;
	while(length>SYNC_LEN){
		POST("Iteration number",a+1);
		CEVENTSET(start,stop,ctime);
		row_kernel<<<grid,block>>>(arr[pos%2],arr[(pos+1)%2]);
		CEVENTGET(start,stop,ctime);
		*rtot=*rtot+(double)ctime;
		err = hipGetLastError();
		if(err!=hipSuccess){
			cout<<err<<endl;
			cout<<length<<","<<grid.x<<endl;
			cout<<a+1<<endl;
			return err;
		}
		pos++;
		length = grid.x;
		lag = (1024-length%1024)%1024;
		if(length<=SYNC_LEN || (lag>LAG_THRESH  && (length-length%1024)<=SYNC_LEN))break;
		else if(length%1024!=0){
			if(lag<LAG_THRESH){
				length = length+lag;
				MSG("Length lag < threshold. Adjusting with memset");
				CEVENTSET(start,stop,ctime);
				err = hipMemset(arr[pos%2]+MAT_SIZE*grid.x,0,MAT_SIZE*sizeof(double)*(lag));
				CEVENTGET(start,stop,ctime);
				*rtot=*rtot+(double)ctime;
				CDEBUG("Memset during length lag adjustment",err);
				grid.x = length/1024;
			}
			else{
				lag = length%1024;
				length-=lag;
				double *temp = new double[(lag+1)*MAT_SIZE];
				double *temp_ans = new double[MAT_SIZE];
				MSG("Length lag>=threshold. Adjusting with collapsing");
				CEVENTSET(start,stop,ctime);
				err = hipMemcpy(temp,arr[pos%2]+MAT_SIZE*(length-1),MAT_SIZE*sizeof(double)*(lag+1),hipMemcpyDeviceToHost);
				CEVENTGET(start,stop,ctime);
				*mtot=*mtot+(double)ctime;
				CDEBUG("Memcpy during length lag collapse",err);
				push_clock();
				lowRedux(temp,lag+1,temp_ans,flag);
				diff = push_clock();
				*rtot = *rtot + diff;
				MSG("Copying back after collapse");
				CEVENTSET(start,stop,ctime);
				err = hipMemcpy(arr[pos%2]+MAT_SIZE*(length-1),temp_ans,MAT_SIZE*sizeof(double),hipMemcpyHostToDevice);
				CEVENTGET(start,stop,ctime);
				*mtot=*mtot+(double)ctime;
				CDEBUG("Memcpy after collapse",err);
				delete[] temp;
				delete[] temp_ans;
				grid.x = length/1024;
			}
		}			
		else grid.x/=1024;
		++a;
	}
	clear_clock();
	return getRowResult(arr[pos%2],length,bans,flag,rtot,mtot);
}
hipError_t getRowResult(double *d_omat,size_t length,vector<double> &bans,int flag,double *rtot,double *mtot){
	double *h_mat = new double[MAT_SIZE*length];
	hipError_t err;
	double diff;
	hipEvent_t start,stop;
	float ctime=0;
	CEVENTSET(start,stop,ctime);
	err = hipMemcpy(h_mat,d_omat,MAT_SIZE*length*sizeof(double),hipMemcpyDeviceToHost);
	CEVENTGET(start,stop,ctime);
	*mtot=*mtot+(double)ctime;
	if(err!=hipSuccess)return err;
	double *h_ans = new double[MAT_SIZE];
	push_clock();
	lowRedux(h_mat,length,h_ans,flag);
	diff = push_clock();
	*rtot = *rtot + diff;
	bans.push_back(h_ans[0]);
	bans.push_back(h_ans[1]);
	bans.push_back(h_ans[2]);
	bans.push_back(h_ans[3]);
	delete[] h_mat;
	delete[] h_ans;
	clear_clock();
	return hipSuccess;
}
hipError_t colRedux(dim3 grid,dim3 block,double *d_imat,double *d_omat,size_t length,vector<double> &bans,int flag,double *rtot,double *mtot){
	double *arr[]={d_imat,d_omat};
	int pos=0;
	hipError_t err;
	POST("Reducing on length",length);
	POST("Sync length",SYNC_LEN);
	float ctime;
	double diff;
	hipEvent_t start,stop;
	unsigned int lag;
	int a=0;
	while(length>SYNC_LEN){
		POST("Iteration number",a+1);
		CEVENTSET(start,stop,ctime);
		col_kernel<<<grid,block>>>(arr[pos%2],arr[(pos+1)%2],length);
		CEVENTGET(start,stop,ctime);
		*rtot=*rtot+(double)ctime;
		err = hipGetLastError();
		if(err!=hipSuccess){
			cout<<err<<endl;
			cout<<length<<","<<grid.x<<endl;
			cout<<a+1<<endl;
			return err;
		}
		pos++;
		length = grid.x;
		if(length<=SYNC_LEN ||  (length-length%1024)<=SYNC_LEN)break;
		else if(length%1024!=0){
			/**lag = length%1024;
			length-=lag;
			double *temp = new double[(lag+1)*MAT_SIZE];
			double *temp_ans = new double[MAT_SIZE];
			MSG("Length lag>=threshold. Adjusting with collapsing");
			CEVENTSET(start,stop,ctime);
			err = hipMemcpy(temp,arr[pos%2]+MAT_SIZE*(length-1),MAT_SIZE*sizeof(double)*(lag+1),hipMemcpyDeviceToHost);
			CEVENTGET(start,stop,ctime);
			*mtot=*mtot+(double)ctime;
			CDEBUG("Memcpy during length lag collapse",err);
			push_clock();
			lowRedux(temp,lag+1,temp_ans,flag);
			diff = push_clock();
			*rtot = *rtot + diff;
			MSG("Copying back after collapse");
			CEVENTSET(start,stop,ctime);
			err = hipMemcpy(arr[pos%2]+MAT_SIZE*(length-1),temp_ans,MAT_SIZE*sizeof(double),hipMemcpyHostToDevice);
			CEVENTGET(start,stop,ctime);
			*mtot=*mtot+(double)ctime;
			CDEBUG("Memcpy after collapse",err);
			delete[] temp;
			delete[] temp_ans;
			grid.x = length/1024;*/
		}		
		else grid.x/=1024;
		++a;
	}
	clear_clock();
	return getColResult(arr[pos%2],length,bans,flag,rtot,mtot);
}
hipError_t getColResult(double *d_omat,size_t length,vector<double> &bans,int flag,double *rtot,double *mtot){
	double *h_mat = new double[MAT_SIZE*length];
	hipError_t err;
	double diff;
	hipEvent_t start,stop;
	float ctime=0;
	CEVENTSET(start,stop,ctime);
	err = hipMemcpy(h_mat,d_omat,MAT_SIZE*length*sizeof(double),hipMemcpyDeviceToHost);
	CEVENTGET(start,stop,ctime);
	*mtot=*mtot+(double)ctime;
	if(err!=hipSuccess)return err;
	double *h_ans = new double[MAT_SIZE];
	push_clock();
	lowRedux(h_mat,length,h_ans,flag);
	diff = push_clock();
	*rtot = *rtot + diff;
	bans.push_back(h_ans[0]);
	bans.push_back(h_ans[1]);
	bans.push_back(h_ans[2]);
	bans.push_back(h_ans[3]);
	delete[] h_mat;
	delete[] h_ans;
	clear_clock();
	return hipSuccess;
}
void addMat(vector<double> &bans,vector<double> &ans,int flag){
	int l = bans.size()/4;
	ans[0]=ans[1]=ans[2]=ans[3]=0.0;
	if(!flag){
		for(int a=0;a<l;a++){
			ans[0]+=bans[a*4];
			ans[1]+=bans[a*4+1];
			ans[2]+=bans[a*4+2];
			ans[3]+=bans[a*4+3];
		}
	}
}
