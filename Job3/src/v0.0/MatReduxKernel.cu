#include "hip/hip_runtime.h"
#include "kernelRedux.h"
__global__ void  row_kernel(double *imat , double *omat,size_t grids){
	__shared__ double  sdata [1024*4];
	unsigned  int tid = 4*threadIdx.x;
	unsigned  int i = 4*(blockIdx.x*blockDim.x + threadIdx.x);
	sdata[tid] = imat[i];
	sdata[tid+1] = imat[i+1];
	sdata[tid+2] = imat[i+2];
	sdata[tid+3] = imat[i+3];
	__syncthreads();
	for(unsigned  int s=4;s<4*blockDim.x;s*= 2)
	{
		if (tid  %(2*s)==0){
			sdata[tid]+= sdata[tid+s];
			sdata[tid+1]+=sdata[tid+1+s];
			sdata[tid+2]+=sdata[tid+2+s];
			sdata[tid+3]+=sdata[tid+3+s];
		}
		__syncthreads();
	}
	if (tid ==0){
		unsigned int bid = 4*blockIdx.x;
		omat[bid] = sdata [0];
		omat[bid+1] = sdata[1];
		omat[bid+2] = sdata[2];
		omat[bid+3] = sdata[3];
	}
}
__global__ void  col_kernel(double *imat , double *omat,size_t length){
	__shared__ double  sdata [1024*4];
	unsigned  int tid = threadIdx.x;
	unsigned  int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = imat[i];
	sdata[tid+1*1024] = imat[i+1*length];
	sdata[tid+2*1024] = imat[i+2*length];
	sdata[tid+3*1024] = imat[i+3*length];
	__syncthreads();
	for(unsigned  int s=1;s<blockDim.x;s*= 2)
	{
		if (tid  %(2*s)==0){
			sdata[tid]+= sdata[tid+s];
			sdata[tid+1*1024]+=sdata[tid+1*1024+s];
			sdata[tid+2*1024]+=sdata[tid+2*1024+s];
			sdata[tid+3*1024]+=sdata[tid+3*1024+s];
		}
		__syncthreads();
	}
	if (tid ==0){
		unsigned int bid = blockIdx.x;
		omat[bid] = sdata [0];
		omat[bid+1*length/1024] = sdata[1*1024];
		omat[bid+2*length/1024] = sdata[2*1024];
		omat[bid+3*length/1024] = sdata[3*1024];
	}
}
