#include "hip/hip_runtime.h"
#include "kernelRedux.h"
__global__ void  row_kernel(double *imat , double *omat,size_t grids){
	unsigned  int tid = 4*threadIdx.x;
	unsigned  int i = 4*(blockIdx.x*blockDim.x*2 + threadIdx.x);
	if(blockIdx.x*2+1==grids)return;
	__shared__ double  sdata [1024*MAT_SIZE];
	if(blockIdx.x*2+3==grids){
		sdata[tid] = imat[i] + imat[i+4*blockDim.x] + imat[i+8*blockDim.x];
		sdata[tid+1] = imat[i+1] + imat[i+4*blockDim.x+1] + imat[i+8*blockDim.x+1];
		sdata[tid+2] = imat[i+2] + imat[i+4*blockDim.x+2] + imat[i+8*blockDim.x+2];
		sdata[tid+3] = imat[i+3] + imat[i+4*blockDim.x+3] + imat[i+8*blockDim.x+3];

	}
	else{	
		sdata[tid] = imat[i]+imat[i+4*blockDim.x];
		sdata[tid+1] = imat[i+1] + imat[i+4*blockDim.x+1];
		sdata[tid+2] = imat[i+2] + imat[i+4*blockDim.x+2];
		sdata[tid+3] = imat[i+3] + imat[i+4*blockDim.x+3];
	}
	__syncthreads();
	IFREDOP(sdata,tid,4*512);
	IFREDOP(sdata,tid,4*256);
	IFREDOP(sdata,tid,4*128);
	IFREDOP(sdata,tid,4*64);
	if(tid<4*32){
		REDOP(sdata,tid,4*32);
		//__syncthreads();
		//__threadfence();
		__threadfence_block();
		//__syncwarp(0xA);
		REDOP(sdata,tid,4*16);
		//__syncthreads();
		//__threadfence();
		__threadfence_block();
		//__syncwarp(0xA);
		REDOP(sdata,tid,4*8);
		//__syncthreads();
		//__threadfence();
		__threadfence_block();
		//__syncwarp(0xA);
		REDOP(sdata,tid,4*4);
		//__syncthreads();
		//__threadfence();
		__threadfence_block();
		//__syncwarp(0xA);
		REDOP(sdata,tid,4*2);
		//__syncthreads();
		//__threadfence();
		__threadfence_block();
		//__syncwarp(0xA);
		REDOP(sdata,tid,4*1);
		if (tid ==0){
			unsigned int bid = 4*blockIdx.x;
			omat[bid] = sdata [0];
			omat[bid+1] = sdata[1];
			omat[bid+2] = sdata[2];
			omat[bid+3] = sdata[3];
		}
	}
}
__global__ void  col_kernel(double *imat , double *omat,size_t length){
	__shared__ double  sdata [1024*MAT_SIZE];
	unsigned  int tid = threadIdx.x;
	unsigned  int i = blockIdx.x*blockDim.x*2 + threadIdx.x;
	if(blockIdx.x*2+1>=gridDim.x)return;
	else if(blockIdx.x*2+3==gridDim.x){
		sdata[tid+0*1024] = imat[i+0*length] + imat[i+blockDim.x+0*length] + imat[i+2*blockDim.x+0*length];
		sdata[tid+1*1024] = imat[i+1*length] + imat[i+blockDim.x+1*length] + imat[i+2*blockDim.x+1*length];
		sdata[tid+2*1024] = imat[i+2*length] + imat[i+blockDim.x+2*length] + imat[i+2*blockDim.x+2*length];
		sdata[tid+3*1024] = imat[i+3*length] + imat[i+blockDim.x+3*length] + imat[i+2*blockDim.x+3*length];

	}
	else{	
		sdata[tid+0*1024] = imat[i+0*length] + imat[i+blockDim.x+0*length];
		sdata[tid+1*1024] = imat[i+1*length] + imat[i+blockDim.x+1*length];
		sdata[tid+2*1024] = imat[i+2*length] + imat[i+blockDim.x+2*length];
		sdata[tid+3*1024] = imat[i+3*length] + imat[i+blockDim.x+3*length];
	}
	__syncthreads();
	IFREDOPCOL(sdata,tid,512);
	IFREDOPCOL(sdata,tid,256);
	IFREDOPCOL(sdata,tid,128);
	IFREDOPCOL(sdata,tid,64);
	if(tid<4*32){
		REDOPCOL(sdata,tid,32);
		//__syncthreads();
		//__threadfence();
		__threadfence_block();
		//__syncwarp(0xA);
		REDOPCOL(sdata,tid,16);
		//__syncthreads();
		//__threadfence();
		__threadfence_block();
		//__syncwarp(0xA);
		REDOPCOL(sdata,tid,8);
		//__syncthreads();
		//__threadfence();
		__threadfence_block();
		//__syncwarp(0xA);
		REDOPCOL(sdata,tid,4);
		//__syncthreads();
		//__threadfence();
		__threadfence_block();
		//__syncwarp(0xA);
		REDOPCOL(sdata,tid,2);
		//__syncthreads();
		//__threadfence();
		__threadfence_block();
		//__syncwarp(0xA);
		REDOPCOL(sdata,tid,1);
		if (tid ==0){
			unsigned int bid = blockIdx.x;
			omat[bid] = sdata [0];
			omat[bid+1*length/1024] = sdata[1*1024];
			omat[bid+2*length/1024] = sdata[2*1024];
			omat[bid+3*length/1024] = sdata[3*1024];
		}
	}
}
