#include "hip/hip_runtime.h"
#include "kernelRedux.h"
__global__ void  row_kernel(double *imat , double *omat){
	__shared__ double  sdata [1024*4];
	unsigned  int tid = 4*threadIdx.x;
	unsigned  int i = 4*(blockIdx.x*blockDim.x*2 + threadIdx.x);
	if(blockIdx.x*2+1>=gridDim.x)return;
	else if(blockIdx.x*2+3==gridDim.x){
		sdata[tid] = imat[i] + imat[i+4*blockDim.x] + imat[i+8*blockDim.x];
		sdata[tid+1] = imat[i+1] + imat[i+4*blockDim.x+1] + imat[i+8*blockDim.x+1];
		sdata[tid+2] = imat[i+2] + imat[i+4*blockDim.x+2] + imat[i+8*blockDim.x+2];
		sdata[tid+3] = imat[i+3] + imat[i+4*blockDim.x+3] + imat[i+8*blockDim.x+3];

	}
	else{	
		sdata[tid] = imat[i]+imat[i+4*blockDim.x];
		sdata[tid+1] = imat[i+1] + imat[i+4*blockDim.x+1];
		sdata[tid+2] = imat[i+2] + imat[i+4*blockDim.x+2];
		sdata[tid+3] = imat[i+3] + imat[i+4*blockDim.x+3];
	}
	__syncthreads();
	IFREDOP(sdata,tid,4*512);
	IFREDOP(sdata,tid,4*256);
	IFREDOP(sdata,tid,4*128);
	IFREDOP(sdata,tid,4*64);
	if(tid<4*32){
		REDOP(sdata,tid,4*32);
		//__syncthreads();
		//__threadfence();
		__threadfence_block();
		//__syncwarp(0xA);
		REDOP(sdata,tid,4*16);
		//__syncthreads();
		//__threadfence();
		__threadfence_block();
		//__syncwarp(0xA);
		REDOP(sdata,tid,4*8);
		//__syncthreads();
		//__threadfence();
		__threadfence_block();
		//__syncwarp(0xA);
		REDOP(sdata,tid,4*4);
		//__syncthreads();
		//__threadfence();
		__threadfence_block();
		//__syncwarp(0xA);
		REDOP(sdata,tid,4*2);
		//__syncthreads();
		//__threadfence();
		__threadfence_block();
		//__syncwarp(0xA);
		REDOP(sdata,tid,4*1);
		if (tid ==0){
			unsigned int bid = 4*blockIdx.x;
			omat[bid] = sdata [0];
			omat[bid+1] = sdata[1];
			omat[bid+2] = sdata[2];
			omat[bid+3] = sdata[3];
		}
	}
}
