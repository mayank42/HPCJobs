#include "hip/hip_runtime.h"
#include "kernelRedux.h"
__global__ void  row_kernel(double *imat , double *omat){
	__shared__ double  sdata [1024*4];
	unsigned  int tid = 4*threadIdx.x;
	unsigned  int i = 4*(blockIdx.x*blockDim.x + threadIdx.x);
	sdata[tid] = imat[i];
	sdata[tid+1] = imat[i+1];
	sdata[tid+2] = imat[i+2];
	sdata[tid+3] = imat[i+3];
	__syncthreads();
	for(unsigned  int s=2*blockDim.x;s>=4;s>>=1)
	{
		if (tid<s){
			sdata[tid]+= sdata[tid+s];
			sdata[tid+1]+=sdata[tid+1+s];
			sdata[tid+2]+=sdata[tid+2+s];
			sdata[tid+3]+=sdata[tid+3+s];
		}
		__syncthreads();
	}
	if (tid ==0){
		unsigned int bid = 4*blockIdx.x;
		omat[bid] = sdata [0];
		omat[bid+1] = sdata[1];
		omat[bid+2] = sdata[2];
		omat[bid+3] = sdata[3];
	}
}
