#include "hip/hip_runtime.h"
#include "kernelRedux.h"
__global__ void  row_kernel(double *imat , double *omat,size_t grids){
	__shared__ double  sdata [1024*4];
	unsigned  int tid = 4*threadIdx.x;
	unsigned  int i = 4*(blockIdx.x*blockDim.x + threadIdx.x);
	sdata[tid] = imat[i];
	sdata[tid+1] = imat[i+1];
	sdata[tid+2] = imat[i+2];
	sdata[tid+3] = imat[i+3];
	tid/=4;
	__syncthreads();
	unsigned int index;
	for(unsigned  int s=4;s<4*blockDim.x;s<<=1)
	{
		index = 2*s*tid;
		if (index<4*blockDim.x){
			sdata[index]+= sdata[index+s];
			sdata[index+1]+=sdata[index+1+s];
			sdata[index+2]+=sdata[index+2+s];
			sdata[index+3]+=sdata[index+3+s];
		}
		__syncthreads();
	}
	if (tid ==0){
		unsigned int bid = 4*blockIdx.x;
		omat[bid] = sdata [0];
		omat[bid+1] = sdata[1];
		omat[bid+2] = sdata[2];
		omat[bid+3] = sdata[3];
	}
}
